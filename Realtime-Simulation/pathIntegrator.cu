#include "hip/hip_runtime.h"
/*
*  pathIntegrator.cu
*  Copyright (c) 2014 Travis Johnson
*  Distributed under the MIT License
*/

#include <fstream>
#include <iostream>
#include <iomanip>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>
#include <cula.h>

#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/reduce.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>

/* An attempt to make the code compile on Apple computers as well */
#if defined(__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#include <OpenGL/gl.h>
#include <OpenGL/glu.h>
#else
#include <GL/glew.h>
#include <GL/gl.h>
#include <GL/freeglut.h>
#endif
#include <cuda_gl_interop.h>

#include "my_helper.h"

#define OPENGL 0
#define ENERGY 1
#define NUMSTATS 5

static const unsigned int NUM_D = 1024;
static const unsigned int MAT_SIZE = NUM_D*NUM_D;
static const double XMIN = -6; // Shoule be no less than -8
static const double XMAX = 6; // Shoule be no more than 8
static const double DELTAX = (XMAX - XMIN)/NUM_D;

static const double PI = 3.141592653589793;
static const double PERIOD = 2*PI;

static const unsigned int NUM_T = 256; //256 works alright
static const double EPST = PERIOD/NUM_T;
static const unsigned int EPST_PER_STEP = 1;

static const unsigned int WARPSIZE = 32;


__device__ __constant__ double NUM_D_d = NUM_D;
__device__ __constant__ double XMIN_d = XMIN;
__device__ __constant__ double DELTAX_d = DELTAX;
__device__ __constant__ double PI_d = PI;

// Some global variables for openGL
GLuint gl_vbo;
GLuint gl_vboPotential;
GLuint gl_program;
GLint attrib_coord2d;
GLint uniform_color;
GLint uniform_xscale;
struct hipGraphicsResource* cuda_vbo_resource; 

// Global pointers for host data
double* x_grid_h;
hipDoubleComplex *wavefunction_h;
hipDoubleComplex *propagator_h;

double *statistics;

// Global Pointers for device data
double* x_grid_d;
hipDoubleComplex *wavefunction_d;
hipDoubleComplex *infintesimal_propagator_d;
hipDoubleComplex *propagator_d;

// Handle to the cublas context that we will create
hipblasHandle_t cublasHandle;
int windowID;

///////////////////////////////////////////////////////////////////////////////
// Function prototype forward declarations
///////////////////////////////////////////////////////////////////////////////

void allocateMemory();
void cleanUp();

// CUDA kernels
__global__ void genXGrid(double*);
__global__ void genInitialWavefunction(hipDoubleComplex*, const double*);
__global__ void genInfintesimalPropagator(hipDoubleComplex*, const double*);
__host__ __device__ __inline__ double PotentialEnergy(double);
__host__ __device__ __inline__ hipDoubleComplex complexExponential(double);

// Host routines
void genPropagator(hipDoubleComplex*, hipDoubleComplex*, unsigned int);
void applyPropagator(hipDoubleComplex*, hipDoubleComplex*);
void computeStatistics(double*, double*, hipDoubleComplex*);
void printPropagator();

// Now for opengl and glut
void initGL(int argc, char** argv);
GLuint createProgram();
void display(void);
void keyboard(unsigned char, int, int);
__global__ void genPotentialVBO(float2 *vbo, double *x_grid);
__global__ void updateVBO(float2 *vbo_d, double *x_grid_d, hipDoubleComplex *wavefunction_d);

// Useful Error checking code
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

void allocateMemory()
{
    // Allocate Memory
    wavefunction_h = (hipDoubleComplex*) malloc (NUM_D * sizeof(hipDoubleComplex));
    propagator_h = (hipDoubleComplex*) malloc (MAT_SIZE * sizeof(hipDoubleComplex));
    x_grid_h = (double*) malloc (NUM_D * sizeof(double));

    hipMalloc (&x_grid_d, NUM_D * sizeof(double)); 
    hipMalloc (&wavefunction_d, NUM_D * sizeof(hipDoubleComplex)); 
    hipMalloc (&infintesimal_propagator_d, MAT_SIZE * sizeof(hipDoubleComplex)); 
    hipMalloc (&propagator_d, MAT_SIZE * sizeof(hipDoubleComplex)); 

    statistics = (double*) malloc (NUMSTATS * sizeof(double));

    // We will use cublas to handle all the cuda stuff, so we
    // need to initialize a cublas context first.
    if (hipblasCreate(&cublasHandle) != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "Could not create cublas handle.\n");
    }
}

void cleanUp()
{
    hipblasDestroy(cublasHandle);

    hipGraphicsUnregisterResource(cuda_vbo_resource);
    glDeleteBuffers(1, &gl_vbo);
    glDeleteBuffers(1, &gl_vboPotential);

    free (wavefunction_h);
    free (x_grid_h);
    free (propagator_h);
    free (statistics);
    hipFree(x_grid_d);
    hipFree(wavefunction_d);
    hipFree(infintesimal_propagator_d);

    hipDeviceReset();
}

// Function to compute the potential energy on the device
__host__ __device__ __inline__ double PotentialEnergy(double x)
{
    double a4 = 1.0;
    double a2 = -2.0;
    double offset = a2*a2/(4*a4);
    double xsquared = x*x;

    return 0.25*a4*xsquared*xsquared + 0.5*a2*xsquared + offset; // for a double well
    //return .5*x*x; // for a SHO
}

///////////////////////////////////////////////////////////////////////////////
// Functions to generate the initial conditions for the problem on the GPU
///////////////////////////////////////////////////////////////////////////////
__global__ void genXGrid(double *grid)
{
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    grid[i] = XMIN_d + i*DELTAX_d;
}

__global__ void genInitialWavefunction(hipDoubleComplex *result, const double *grid)
{
    const double x_start = 1.414214;
    const double alpha = 4;

    const double norm = pow(alpha/PI_d, .25);
    const double factor = alpha/2;
    double dist;

    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

    dist = grid[i] - x_start;
    result[i] = make_hipDoubleComplex( norm * exp(-factor* dist*dist ), 0 );
}


// Returns the complex solution of e^(i*x) as a hipDoubleComplex
__host__ __device__ __inline__ hipDoubleComplex complexExponential(double x)
{
    return make_hipDoubleComplex( cos(x), sin(x) );
}

__global__ void genInfintesimalPropagator(hipDoubleComplex *result, const double *grid)
{
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;

    double tmp = sqrt(1/(4*PI_d*EPST));
    hipDoubleComplex Anorm = make_hipDoubleComplex( tmp, -tmp); 
    hipDoubleComplex tmpC;

    tmp = (grid[i]-grid[j]) / EPST; // the velocity
    // Uses the inline function defined above
    /*
     *tmpC = complexExponential( EPST * ( 0.5 * tmp*tmp
     *                                     //- PotentialEnergy( 0.5*(grid[i]+grid[j]) ) )
     *                                     - 0.5 * (PotentialEnergy(grid[i])+ PotentialEnergy(grid[j]))  )
     *                                     //- PotentialEnergy(grid[i])  )
     *                                    );
     */
    tmpC = complexExponential( EPST * 0.5 * tmp*tmp );
    tmpC = hipCmul(tmpC, complexExponential( -EPST * /*0.5 */ PotentialEnergy( grid[i] ) ) );

    *(result + j*NUM_D + i) = hipCmul(Anorm, tmpC);
}

__global__ void genAnalyticPropagator(hipDoubleComplex *result, const double *grid, double time)
{
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;

    double tmp = sqrt(1/(4*PI_d*sin(time)));
    hipDoubleComplex Anorm = make_hipDoubleComplex( tmp, -tmp); 
    hipDoubleComplex tmpC;

    tmpC = complexExponential( 1/(2*sin(time)) *( ( grid[j]*grid[j] + grid[i]*grid[i] ) * cos(time)
                                         - 2*grid[i]*grid[j]));

    *(result + j*NUM_D + i) = hipCmul(Anorm, tmpC);
}

// Generate a finite time propagator that is the given power of the infintesimal propagator
void genPropagator(hipDoubleComplex *propagator, hipDoubleComplex *infintesimal_prop, unsigned int power)
{
    const hipDoubleComplex alpha = make_hipDoubleComplex( DELTAX, 0);
    const hipDoubleComplex beta = make_hipDoubleComplex( 0, 0);

    // Quick exit if power is 1
    if (power == 1) {
        hipMemcpy( propagator, infintesimal_prop, MAT_SIZE * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice );
        return;
    }

    // A temporary variable to store intermediate steps
    hipDoubleComplex *temp_d;
    hipMalloc (&temp_d, MAT_SIZE * sizeof(hipDoubleComplex)); 

    // Set the propagator equal to the infintesimal propagator
    hipMemcpy( propagator, infintesimal_prop, MAT_SIZE * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice );

    // We want this code to run power-1 times
    for (int i=1; i < power; i++)
    {
    /*
        hipblasStatus_t hipblasZgemm(hipblasHandle_t handle,
                                hipblasOperation_t transa, hipblasOperation_t transb,
                                int m, int n, int k,
                                const hipDoubleComplex *alpha,
                                const hipDoubleComplex *A, int lda,
                                const hipDoubleComplex *B, int ldb,
                                const hipDoubleComplex *beta,
                                hipDoubleComplex *C, int ldc)
    */
    hipblasZgemm( cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                    NUM_D, NUM_D, NUM_D,
                    &alpha, 
                    infintesimal_prop, NUM_D,
                    propagator, NUM_D,
                    &beta, 
                    temp_d, NUM_D);

    hipMemcpy( propagator, temp_d, MAT_SIZE * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);
    }

    hipFree( temp_d );
}

/*
__global__ void computeTrace( hipDoubleComplex *result_d, const hipDoubleComplex *matrix_d )
{
    const hipDoubleComplex deltaxC = make_hipDoubleComplex( DELTAX_d, 0);
    *result_d = make_hipDoubleComplex(0,0);
    for (unsigned int j = 0; j < NUM_D; j++) {
        *result_d = hipCadd( *result_d, *(matrix_d + j*(NUM_D+1)) );
    }
    // Multiply by deltaxC since this is an integral
    *result_d = hipCmul(*result_d, deltaxC);
}

__global__ void fillTraceVec( hipDoubleComplex *vec_data_d, const hipDoubleComplex *matrix_d )
{
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

    *(vec_data_d+i) = *(matrix_d + i*(NUM_D+1));
}
*/

// Functor for use below with thrust::reduce
struct cuCadd_functor 
{ 
    __host__ __device__ 
        hipDoubleComplex operator()(const hipDoubleComplex& x, const hipDoubleComplex& y) const 
        { 
            return hipCadd( x, y );
        } 
};

// hipCmul functor to be used with thrust::transform
struct cuCmul_functor 
{ 
    __host__ __device__ 
        hipDoubleComplex operator()(const hipDoubleComplex& x, const hipDoubleComplex& y) const 
        { return hipCmul( x, y ); } 
};

// Computes the trace of the propagator at int*epsilson time steps for int from 1 to numSteps
void computeTraceFFT( hipDoubleComplex *traceFFT_d, hipDoubleComplex *inf_prop_d, unsigned int numSteps )
{
    int step = 0;
    const hipDoubleComplex deltaxC = make_hipDoubleComplex( DELTAX, 0);
    const hipDoubleComplex zeroC = make_hipDoubleComplex( 0, 0);

    // Storage for the trace of the propagator as a function of time
    hipDoubleComplex *trace_h;
    trace_h = (hipDoubleComplex*) malloc( numSteps * sizeof(hipDoubleComplex) );
    // On the device as well for cuFFT
    hipDoubleComplex *trace_d;
    hipMalloc( &trace_d, numSteps * sizeof(hipDoubleComplex) );

    // The input and output matricies from the gemm operation
    // The output will be an epsilon time step later than the input
    hipDoubleComplex *input_prop_d;
    hipDoubleComplex *output_prop_d;
    hipMalloc( &input_prop_d, MAT_SIZE * sizeof(hipDoubleComplex) );
    hipMalloc( &output_prop_d, MAT_SIZE * sizeof(hipDoubleComplex) );

    // Set the input propagator equal to the infintesimal propagator
    hipMemcpy( input_prop_d, inf_prop_d, MAT_SIZE * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice );


    // Fill the trace_d matrix with the time dependence of the trace
    step = 0;
    do {
        // device ptr to the current propagator matrix
        thrust::device_ptr<hipDoubleComplex> prop_ptr( input_prop_d );
        // custom iterator to stride into the array to get only the diagonal elements
        strided_range< thrust::device_vector<hipDoubleComplex>::iterator >
            trace_vec( prop_ptr, prop_ptr + MAT_SIZE, NUM_D + 1 );

        trace_h[step] =  thrust::reduce( trace_vec.begin(), trace_vec.end(), zeroC, cuCadd_functor() );
        trace_h[step] = hipCmul( trace_h[step], deltaxC );

        // Advance the time by multiplying by the infintesimal propagator
        /*  hipblasZgemm calling prototype C = alpha A B + beta C
            hipblasStatus_t hipblasZgemm(hipblasHandle_t handle,
                                    hipblasOperation_t transa, hipblasOperation_t transb,
                                    int m, int n, int k,
                                    const hipDoubleComplex *alpha,
                                    const hipDoubleComplex *A, int lda,
                                    const hipDoubleComplex *B, int ldb,
                                    const hipDoubleComplex *beta,
                                    hipDoubleComplex *C, int ldc)
        */
        hipblasZgemm( cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                        NUM_D, NUM_D, NUM_D,
                        &deltaxC, 
                        inf_prop_d, NUM_D,
                        input_prop_d, NUM_D,
                        &zeroC, 
                        output_prop_d, NUM_D);

        // Swap the pointers for the two prop buffers
        std::swap( input_prop_d, output_prop_d );

        // Increment the step and continue
        step++;
    } while ( step < numSteps );

    // Copy the trace information back to the device for cuFFT
    hipMemcpy( trace_d, trace_h, numSteps * sizeof(hipDoubleComplex), hipMemcpyHostToDevice );

    // Now that we have computed the time dependence of the trace, we want to take the FFT
    hipfftHandle plan;

    if( hipfftPlan1d( &plan, numSteps, HIPFFT_Z2Z, 1) != HIPFFT_SUCCESS )
        std::cout << "cuFFT: Plan creation failed." << std::endl;

    if( hipfftExecZ2Z( plan, trace_d, traceFFT_d, HIPFFT_FORWARD) != HIPFFT_SUCCESS )
        std::cout << "cuFFT: Transformation failed." << std::endl;

    hipDeviceSynchronize();
    hipfftDestroy( plan );

    free( trace_h );
    hipFree( input_prop_d );
    hipFree( output_prop_d );
    hipFree( trace_d );
}

hipDoubleComplex analyticTrace( double *x_grid_h, double time )
{
    hipDoubleComplex trace = make_hipDoubleComplex(0,0);
    double tmp = sqrt(1/(4*PI*sin(time)));

    hipDoubleComplex Anorm = make_hipDoubleComplex( tmp, -tmp); 
    hipDoubleComplex tmpC;

    for( int i = 0; i < NUM_D; i++ ) {
        tmpC = complexExponential( 1/(2*sin(time)) * ( 2*x_grid_h[i]*x_grid_h[i] * (cos(time) - 1) ) );
        tmpC = hipCmul(Anorm, tmpC);

        trace = hipCadd( trace, tmpC);
    }

    return trace;
}

void computeTraceFFTAnalytic( hipDoubleComplex *traceFFT_d, double *x_grid_h, unsigned int numSteps )
{
    // Storage for the trace of the propagator as a function of time
    hipDoubleComplex *trace_h;
    trace_h = (hipDoubleComplex*) malloc( numSteps * sizeof(hipDoubleComplex) );
    // On the device as well for cuFFT
    hipDoubleComplex *trace_d;
    hipMalloc( &trace_d, numSteps * sizeof(hipDoubleComplex) );

    for( int i = 0; i < numSteps; i++ ) {
        trace_h[i] = analyticTrace( x_grid_h, EPST*i );
    }

    // Copy the trace information back to the device for cuFFT
    hipMemcpy( trace_d, trace_h, numSteps * sizeof(hipDoubleComplex), hipMemcpyHostToDevice );

    // Now that we have computed the time dependence of the trace, we want to take the FFT
    hipfftHandle plan;

    if( hipfftPlan1d( &plan, numSteps, HIPFFT_Z2Z, 1) != HIPFFT_SUCCESS )
        std::cout << "cuFFT: Plan creation failed." << std::endl;

    if( hipfftExecZ2Z( plan, trace_d, traceFFT_d, HIPFFT_FORWARD) != HIPFFT_SUCCESS )
        std::cout << "cuFFT: Transformation failed." << std::endl;

    hipDeviceSynchronize();
    hipfftDestroy( plan );

    free( trace_h );
    hipFree( trace_d );
}

// Computes the trace of the propagator at multiple times using the
// eigenvalue trick to remove the need for multiplying matricies
void computeTraceFFT2( hipDoubleComplex *traceFFT_d, hipDoubleComplex *inf_prop_d, unsigned int numSteps )
{
    hipDoubleComplex deltaxC = make_hipDoubleComplex( DELTAX, 0);
    hipDoubleComplex zeroC = make_hipDoubleComplex( 0, 0);

    // Storage for the trace of the propagator as a function of time
    hipDoubleComplex *trace_h;
    trace_h = (hipDoubleComplex*) malloc( numSteps * sizeof(hipDoubleComplex) );
    // On the device as well for cuFFT
    hipDoubleComplex *trace_d;
    hipMalloc( &trace_d, numSteps * sizeof(hipDoubleComplex) );

    // Storage for the eigenvalues of the propagator
    hipDoubleComplex *eigenvalues_d;
    hipMalloc( &eigenvalues_d, NUM_D * sizeof(hipDoubleComplex) );
    // And for the vector of powers of the eigenvalues
    hipDoubleComplex *eigen_powers_d;
    hipMalloc( &eigen_powers_d, NUM_D * sizeof(hipDoubleComplex) );
    
    // Use cula library to compute the eigenvalues of the propagator
    culaStatus status = culaInitialize();
    if( status != culaNoError ) {
        printf("%s\n", culaGetStatusString(status));
    }

    status = culaDeviceZgeev('N', 'N', NUM_D, (culaDoubleComplex*) inf_prop_d, NUM_D, (culaDoubleComplex*) eigenvalues_d, NULL, 1, NULL, 1);
    if( status != culaNoError ) {
        if( status == culaDataError )
            printf("Data error with code %d, please see LAPACK documentation\n",culaGetErrorInfo());
        else
            printf("%s\n", culaGetStatusString(status));
    }
    // That's all for cula
    culaShutdown();

    // Duplicate the eigenvalues for the first step in the loop below
    hipMemcpy( eigen_powers_d, eigenvalues_d, NUM_D * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice );

    thrust::device_ptr<hipDoubleComplex> eigen_powers_ptr( eigen_powers_d );
    thrust::device_ptr<hipDoubleComplex> eigenvalues_ptr( eigenvalues_d );

    thrust::constant_iterator<hipDoubleComplex> deltaxC_iter(deltaxC);

    // Sum the eigen_powers vector to get the trace of the matrix
    //  The first element is just the sum of the eigenvalues
    trace_h[0] = thrust::reduce( eigen_powers_ptr, eigen_powers_ptr + NUM_D, zeroC, cuCadd_functor() );
    trace_h[0] = hipCmul( trace_h[0], deltaxC );
    for( int i = 1; i < numSteps; i++)
    {
        // Multiply the current trace vector by the eigenvalues to get (lambda_i)^(i+1)
        thrust::transform( eigen_powers_ptr, eigen_powers_ptr + NUM_D, eigenvalues_ptr, 
                /*output*/ eigen_powers_ptr, cuCmul_functor() );
        // Multiply all elements by deltaxC since the above operation is a simplified
        //  form of the integration
        thrust::transform( eigen_powers_ptr, eigen_powers_ptr + NUM_D, deltaxC_iter, 
                /*output*/ eigen_powers_ptr, cuCmul_functor() );

        // Sum the eigen_powers vector to get the trace of the matrix
        trace_h[i] =  thrust::reduce( eigen_powers_ptr, eigen_powers_ptr + NUM_D, zeroC, cuCadd_functor() );
        // Multiply by deltaX since this was an integration
        trace_h[i] = hipCmul( trace_h[i], deltaxC );
    }

    // Copy the trace information back to the device for cuFFT
    hipMemcpy( trace_d, trace_h, numSteps * sizeof(hipDoubleComplex), hipMemcpyHostToDevice );
    fprintCudaVector( std::cout, trace_d, numSteps );
    

    // Now that we have computed the time dependence of the trace, we want to take the FFT
    hipfftHandle plan;

    if( hipfftPlan1d( &plan, numSteps, HIPFFT_Z2Z, 1) != HIPFFT_SUCCESS )
        std::cout << "cuFFT: Plan creation failed." << std::endl;

    if( hipfftExecZ2Z( plan, trace_d, traceFFT_d, HIPFFT_FORWARD) != HIPFFT_SUCCESS )
        std::cout << "cuFFT: Transformation failed." << std::endl;

    hipDeviceSynchronize();
    hipfftDestroy( plan );

    free( trace_h );
    hipFree( trace_d );
    hipFree( eigenvalues_d );
}

void computeTrace( hipDoubleComplex *result_d, hipDoubleComplex *inf_prop_d, unsigned int numSteps )
{
    if( result_d == NULL ) {
        hipMalloc( &result_d, numSteps * sizeof(hipDoubleComplex) );
    }

    hipDoubleComplex deltaxC = make_hipDoubleComplex( DELTAX, 0);
    hipDoubleComplex zeroC = make_hipDoubleComplex( 0, 0);

    // Storage for the trace of the propagator as a function of time
    hipDoubleComplex *trace_h;
    trace_h = (hipDoubleComplex*) malloc( numSteps * sizeof(hipDoubleComplex) );

    // Storage for the eigenvalues of the propagator
    hipDoubleComplex *eigenvalues_d;
    hipMalloc( &eigenvalues_d, NUM_D * sizeof(hipDoubleComplex) );
    // And for the vector of powers of the eigenvalues
    hipDoubleComplex *eigen_powers_d;
    hipMalloc( &eigen_powers_d, NUM_D * sizeof(hipDoubleComplex) );
    
    // Use cula library to compute the eigenvalues of the propagator
    culaStatus status = culaInitialize();
    if( status != culaNoError ) {
        printf("cula failed to initialize: %s\n", culaGetStatusString(status));
    }

    status = culaDeviceZgeev('N', 'N', NUM_D, (culaDoubleComplex*) inf_prop_d, NUM_D, (culaDoubleComplex*) eigenvalues_d, NULL, 1, NULL, 1);
    if( status != culaNoError ) {
        if( status == culaDataError )
            printf("Data error with code %d, please see LAPACK documentation\n",culaGetErrorInfo());
        else
            printf("%s\n", culaGetStatusString(status));
    }
    // That's all for cula
    culaShutdown();

    // Duplicate the eigenvalues for the first step in the loop below
    hipMemcpy( eigen_powers_d, eigenvalues_d, NUM_D * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice );

    thrust::device_ptr<hipDoubleComplex> eigen_powers_ptr( eigen_powers_d );
    thrust::device_ptr<hipDoubleComplex> eigenvalues_ptr( eigenvalues_d );

    thrust::constant_iterator<hipDoubleComplex> deltaxC_iter(deltaxC);

    // Sum the eigen_powers vector to get the trace of the matrix
    //  The first element is just the sum of the eigenvalues
    trace_h[0] = thrust::reduce( eigen_powers_ptr, eigen_powers_ptr + NUM_D, zeroC, cuCadd_functor() );
    trace_h[0] = hipCmul( trace_h[0], deltaxC );
    for( int i = 1; i < numSteps; i++)
    {
        // Multiply the current trace vector by the eigenvalues to get (lambda_i)^(i+1)
        thrust::transform( eigen_powers_ptr, eigen_powers_ptr + NUM_D, eigenvalues_ptr, 
                /*output*/ eigen_powers_ptr, cuCmul_functor() );
        // Multiply all elements by deltaxC since the above operation is a simplified
        //  form of the integration
        thrust::transform( eigen_powers_ptr, eigen_powers_ptr + NUM_D, deltaxC_iter, 
                /*output*/ eigen_powers_ptr, cuCmul_functor() );

        // Sum the eigen_powers vector to get the trace of the matrix
        trace_h[i] =  thrust::reduce( eigen_powers_ptr, eigen_powers_ptr + NUM_D, zeroC, cuCadd_functor() );
        // Multiply by deltaX since this was an integration
        trace_h[i] = hipCmul( trace_h[i], deltaxC );
    }

    // Copy the trace information back to the device to be returned by this function
    hipMemcpy( result_d, trace_h, numSteps * sizeof(hipDoubleComplex), hipMemcpyHostToDevice );

    free( trace_h );
    hipFree( eigenvalues_d );
    hipFree( eigen_powers_d );
}

void computeFFT(hipDoubleComplex *result_d, hipDoubleComplex *data_d, size_t length, int direction)
{
    if( result_d == NULL ) {
        hipMalloc( &data_d, length * sizeof(hipDoubleComplex) );
    }

    hipfftHandle plan;

    if( hipfftPlan1d( &plan, length, HIPFFT_Z2Z, 1) != HIPFFT_SUCCESS ) {
       std::cout << "cuFFT: Plan creation failed." << std::endl;
       return;
    }

    if( hipfftExecZ2Z( plan, (hipfftDoubleComplex*)data_d, (hipfftDoubleComplex*)result_d, direction) != HIPFFT_SUCCESS ) {
        std::cout << "cuFFT: Transformation failed." << std::endl;
    }

    hipDeviceSynchronize();
    hipfftDestroy( plan );
}

// Apply the propagator matrix to the wavefunction
//TODO Might be better to have a pointer to a scratch buffer passed into this function
//      as well so that it doesn't allocate and destroy new memory each time.
void applyPropagator(hipDoubleComplex *wavefunction, hipDoubleComplex *propagator)
{
    const hipDoubleComplex deltaxC = make_hipDoubleComplex( DELTAX, 0);
    const hipDoubleComplex zeroC  = make_hipDoubleComplex( 0, 0);
    // A temporary variable to store intermediate steps
    hipDoubleComplex *temp_d;
    hipMalloc (&temp_d, NUM_D*sizeof(hipDoubleComplex)); 

    /*
        hipblasStatus_t hipblasZgemv(hipblasHandle_t handle, hipblasOperation_t trans,
                                int m, int n,
                                const hipDoubleComplex *alpha,
                                const hipDoubleComplex *A, int lda,
                                const hipDoubleComplex *x, int incx,
                                const hipDoubleComplex *beta,
                                hipDoubleComplex *y, int incy)
    */

    hipblasZgemv( cublasHandle, HIPBLAS_OP_N,
                    NUM_D, NUM_D,
                    &deltaxC,
                    propagator, NUM_D,
                    wavefunction, 1,
                    &zeroC, 
                    temp_d, 1);

    hipMemcpy( wavefunction, temp_d, NUM_D * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);

    hipFree( temp_d );
}

///////////////////////////////////////////////////////////////////////////////
// OpenGL related functions
///////////////////////////////////////////////////////////////////////////////
void initGL(int argc, char** argv)
{
    GLint window_height = 512;
    GLint window_width = 512;

    glutInit( &argc, argv );
    glutInitDisplayMode( GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize( 512, 512);
    windowID = glutCreateWindow("Wavefunction in a Harmonic Potential");

    std::cout << "Vendor: " << glGetString(GL_VENDOR) << std::endl;
    std::cout << "GL Version: " << glGetString(GL_VERSION) << std::endl;
    // Register callbacks
    glutDisplayFunc( display);
    glutKeyboardFunc( keyboard);

    // Initialize OpenGL extensions with glew if available
#if defined(__APPLE__) || defined(MACOSX)
    // Don't load glew
#else
    glewInit();
#endif

    // Initialize some defaults
    glClearColor(0.0, 0.0, 0.0, 1.0);
    glViewport(0,0, window_width, window_height);

    // Create the vertex buffer objects that we will be using
    glGenBuffers(1, &gl_vbo);
    glBindBuffer(GL_ARRAY_BUFFER, gl_vbo);
    glBufferData(GL_ARRAY_BUFFER, 3*NUM_D*sizeof(float2), NULL, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);

    glGenBuffers(1, &gl_vboPotential);
    glBindBuffer(GL_ARRAY_BUFFER, gl_vboPotential);
    glBufferData(GL_ARRAY_BUFFER, NUM_D*sizeof(float2), NULL, GL_STATIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);

    // Create the program that is used to render the lines to the screen
    gl_program = createProgram();
    attrib_coord2d = glGetAttribLocation( gl_program, "coord2d");
    uniform_color  = glGetUniformLocation( gl_program, "f_color");
    uniform_xscale = glGetUniformLocation( gl_program, "xscale");

    // Default values for uniforms
    glProgramUniform1f(gl_program, uniform_xscale, XMAX);
    glProgramUniform4f(gl_program, uniform_color, 0.0f, 0.0f, 0.0f, 0.0f);

    // Now set up interop between CUDA and OpenGL
    cudaGLSetGLDevice(0);
    // First fill the potential VBO
    float2 *vboPotential_ptr;
    size_t vbo_data_size;
    hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, gl_vboPotential, cudaGraphicsMapFlagsWriteDiscard);
    hipGraphicsMapResources( 1, &cuda_vbo_resource, 0);
    hipGraphicsResourceGetMappedPointer( (void**)&vboPotential_ptr, &vbo_data_size, cuda_vbo_resource);

    genPotentialVBO <<<NUM_D/WARPSIZE, WARPSIZE>>> (vboPotential_ptr, x_grid_d);
    // We will not need access to this resource again
    hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0);
    //hipGraphicsUnregisterResource(

    // Register the wavefunction vbo that will be used for the rest of the program
    hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, gl_vbo, cudaGraphicsMapFlagsWriteDiscard);
}

GLuint createProgram()
{
    const GLchar *vertex_shader_src = 
    {
        "#version 120\n"
        "attribute vec2 coord2d;\n"
        "uniform float xscale;\n"
        //"varying vec4 f_color;\n"
        "\n"
        "void main(void) {\n"
        "   gl_Position = vec4(coord2d.x/xscale, coord2d.y - 0.5, 0, 1);\n"
        //"   f_color = vec4(coord2d.xy / 2.0 + 0.5, 1, 1);\n"
        "}"
    };

    const GLchar *fragment_shader_src = 
    {
        "#version 120\n"
        //"varying vec4 f_color;\n"
        "uniform vec4 f_color;\n"
        "\n"
        "void main(void) {\n"
        "   gl_FragColor = f_color;\n"
        "}"
    };

    GLuint program;
    GLuint Vshader, Fshader;
    GLint check_status = GL_FALSE;
    //const GLchar *vertex_sources[] = {vertex_shader_src};

    program = glCreateProgram();

    Vshader = glCreateShader(GL_VERTEX_SHADER);
    glShaderSource(Vshader, 1, &vertex_shader_src, NULL);
    glCompileShader(Vshader);
    glGetShaderiv(Vshader, GL_COMPILE_STATUS, &check_status);
    if (check_status == GL_FALSE)
    {
        fprintf( stderr, "Vertex shader did not compile.\n");
        return 0;
    }
    glAttachShader(program, Vshader);

    Fshader = glCreateShader(GL_FRAGMENT_SHADER);
    glShaderSource(Fshader, 1, &fragment_shader_src, NULL);
    glCompileShader(Fshader);
    glGetShaderiv(Fshader, GL_COMPILE_STATUS, &check_status);
    if (check_status == GL_FALSE)
    {
        fprintf( stderr, "Fragment shader did not compile.\n");
        return 0;
    }
    glAttachShader(program, Fshader);

    glLinkProgram( program);
    glGetProgramiv( program, GL_LINK_STATUS, &check_status);
    if (check_status == GL_FALSE)
    {
        fprintf( stderr, "Program failed to link.\n");
        return 0;
    }

    return program;
}

__global__ void genPotentialVBO(float2 *vbo, double *x_grid)
{
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

    vbo[i].x = x_grid[i];
    vbo[i].y = PotentialEnergy( x_grid[i] );
}

__global__ void updateVBO(float2 *vbo, double *x_grid, hipDoubleComplex *wavefunction)
{
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    float tmp = hipCabs( wavefunction[i]);

    vbo[i].x = x_grid[i];
    vbo[i].y = tmp*tmp;
}

void display()
{
    float2* vbo_data_d;
    size_t vbo_data_size;

    // Update that data in the VBO
    hipGraphicsMapResources( 1, &cuda_vbo_resource, 0);
    hipGraphicsResourceGetMappedPointer( (void**)&vbo_data_d, &vbo_data_size, cuda_vbo_resource);
    updateVBO <<<NUM_D/WARPSIZE, WARPSIZE>>> (vbo_data_d, x_grid_d, wavefunction_d);
    hipGraphicsUnmapResources( 1, &cuda_vbo_resource, 0);

    // Set every pixel in the frame buffer to the current clear color.
    glClear(GL_COLOR_BUFFER_BIT);

    glEnable(GL_LINE_SMOOTH);
    glUseProgram( gl_program);

    glBindBuffer(GL_ARRAY_BUFFER, gl_vboPotential);
    glEnableVertexAttribArray( attrib_coord2d);
    glVertexAttribPointer( attrib_coord2d, 2, GL_FLOAT, GL_FALSE, 0, 0);
    glLineWidth( 2.0f);
    glProgramUniform4f(gl_program, uniform_color, 1.0f, 0.0f, 0.0f, 1.0f);
    glDrawArrays( GL_LINE_STRIP, 0, NUM_D);

    glBindBuffer(GL_ARRAY_BUFFER, gl_vbo);
    glEnableVertexAttribArray( attrib_coord2d);
    glVertexAttribPointer( attrib_coord2d, 2, GL_FLOAT, GL_FALSE, 0, 0);
    glLineWidth( 2.0f);
    glProgramUniform4f(gl_program, uniform_color, 0.0f, 1.0f, 0.0f, 1.0f);
    glDrawArrays( GL_LINE_STRIP, 0, NUM_D);

    glBindBuffer(GL_ARRAY_BUFFER, 0);
    glDisable(GL_LINE_SMOOTH);

    glutSwapBuffers();
    // Update the system
    applyPropagator( wavefunction_d, propagator_d);

    glutPostRedisplay();
}

void keyboard( unsigned char key, int x, int y)
{
    switch( key) {
    case 32: // Spacebar
        glutPostRedisplay();
        break;
    case 27: // Escape key
        glutDestroyWindow( windowID );
        cleanUp();
        exit(0);
        break;
    }
}

///////////////////////////////////////////////////////////////////////////////
// Global functions to compute statistics for the wavefunction
///////////////////////////////////////////////////////////////////////////////
__global__ void computeNormVec(double *data, double *x_grid, hipDoubleComplex *wavefunction)
{
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    double tmp = hipCabs( wavefunction[i]);
    tmp = tmp*tmp;
    data[i] = tmp * DELTAX_d;
}

__global__ void computeMeanPositionVec(double *data, double *x_grid, hipDoubleComplex *wavefunction)
{
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    double tmp = hipCabs( wavefunction[i]);
    tmp = tmp*tmp;
    data[i] = tmp * x_grid[i] * DELTAX_d;
}

__global__ void computeMeanPotentialEnergyVec(double *data, double *x_grid, hipDoubleComplex *wavefunction)
{
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    double tmp = hipCabs( wavefunction[i]);
    tmp = tmp*tmp;
    data[i] = tmp * PotentialEnergy(x_grid[i]) * DELTAX_d;
}

__global__ void computeMeanKineticEnergyVec(double *data, double *x_grid, hipDoubleComplex *wavefunction)
{
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    hipDoubleComplex tmpComplex;
    double tmp;

    if ( i == NUM_D-1 )
        tmpComplex = hipCsub( wavefunction[0], wavefunction[i] ); // Periodic boundary condition
    else
        tmpComplex = hipCsub( wavefunction[i+1], wavefunction[i] );

    tmp = hipCabs( tmpComplex);
    tmp = tmp*tmp;

    data[i] = 0.5* tmp / DELTAX_d;
}

/*
void computeStatistics(double *stats, double *x_grid, hipDoubleComplex *wavefunction)
{
    unsigned int block_size_1D = WARPSIZE;
    unsigned int grid_size_1D = ceil(NUM_D/WARPSIZE);
    double *data;

    thrust::device_vector<double> vec(NUM_D);
    data = thrust::raw_pointer_cast( vec.data() );

    computeNormVec <<<grid_size_1D, block_size_1D>>>( data, x_grid, wavefunction );
    stats[0] = thrust::reduce( vec.begin(), vec.end(), (double) 0, thrust::plus<double>() );

    computeMeanPositionVec <<<grid_size_1D, block_size_1D>>>( data, x_grid, wavefunction );
    stats[1] = thrust::reduce( vec.begin(), vec.end(), (double) 0, thrust::plus<double>() );

    computeMeanPotentialEnergyVec <<<grid_size_1D, block_size_1D>>>( data, x_grid, wavefunction );
    stats[2] = thrust::reduce( vec.begin(), vec.end(), (double) 0, thrust::plus<double>() );

    computeMeanKineticEnergyVec <<<grid_size_1D, block_size_1D>>>( data, x_grid, wavefunction );
    stats[3] = thrust::reduce( vec.begin(), vec.end(), (double) 0, thrust::plus<double>() );

    stats[4] = stats[2] + stats[3]; // Total energy
}
*/

struct cuCabsSquared_functor
{
    __host__ __device__
        double operator()(const hipDoubleComplex& x) const 
        {
            double tmp = hipCabs(x);
            return tmp*tmp;
        }
};

struct potentialEnergy_functor 
{ 
    __host__ __device__ 
        double operator()(const double& psi_squared, const double& x_pos) const 
        {return psi_squared * PotentialEnergy(x_pos);} 
};

void computeStatistics(double *stats, double *x_grid, hipDoubleComplex *wavefunction)
{
    unsigned int block_size_1D = WARPSIZE;
    unsigned int grid_size_1D = ceil(NUM_D/WARPSIZE);
    thrust::device_ptr<hipDoubleComplex> wavefunction_ptr( wavefunction );
    // device vector to store psi_squared
    thrust::device_vector<double> psi_squared(NUM_D);

    thrust::device_vector<double> vec(NUM_D);
    double *data = thrust::raw_pointer_cast( vec.data() );

    // Compute the absSquared of the wavefunction and store in the vector psi_squared
    thrust::transform( wavefunction_ptr, wavefunction_ptr + NUM_D, psi_squared.begin(), cuCabsSquared_functor() );

    // The norm of the vector from a simple plus reduction
    stats[0] = DELTAX * thrust::reduce( psi_squared.begin(), psi_squared.end(), (double) 0, thrust::plus<double>() );
    // Compute the average position of the particle by multipling the grid value by psi_squared
    thrust::transform( psi_squared.begin(), psi_squared.end(), thrust::device_ptr<double>(x_grid), 
            /*output*/vec.begin(), thrust::multiplies<double>() );
    stats[1] = DELTAX * thrust::reduce( vec.begin(), vec.end(), (double) 0, thrust::plus<double>() );
    // Compute the average potential using the potentialEnergy functor to compute its value at each grid point
    thrust::transform( psi_squared.begin(), psi_squared.end(), thrust::device_ptr<double>(x_grid), 
            /*output*/vec.begin(), potentialEnergy_functor() );
    stats[2] = DELTAX * thrust::reduce( vec.begin(), vec.end(), (double) 0, thrust::plus<double>() );

    // TODO Figure out how to calculate the kinetic energy with just thrust as well
    //  it is a bit harder to deal with the derivatives though
    computeMeanKineticEnergyVec <<<grid_size_1D, block_size_1D>>>( data, x_grid, wavefunction );
    stats[3] = thrust::reduce( vec.begin(), vec.end(), (double) 0, thrust::plus<double>() );

    stats[4] = stats[2] + stats[3]; // Total energy
}

///////////////////////////////////////////////////////////////////////////////
// Main Function
///////////////////////////////////////////////////////////////////////////////
int main(int argc, char *argv[])
{
    size_t blockEdge = 16;
    dim3 block_size;
    dim3 grid_size;

    unsigned int block_size_1D;
    unsigned int grid_size_1D;

    // Allocate all the memory we will use
    allocateMemory();

    hipDoubleComplex *analytic_propagator_d;
    hipMalloc( (void**) &analytic_propagator_d, MAT_SIZE * sizeof(hipDoubleComplex) );

    // Set grid and block sizes
    block_size.x = blockEdge ;
    block_size.y = blockEdge;
    block_size.z = 1;

    grid_size.x = NUM_D/blockEdge;
    grid_size.y = NUM_D/blockEdge;
    grid_size.z = 1;

    block_size_1D = WARPSIZE;
    grid_size_1D = NUM_D/WARPSIZE;

    // Populate the initial arrays
    genXGrid <<<grid_size_1D, block_size_1D>>> (x_grid_d);
    genInitialWavefunction <<<grid_size_1D, block_size_1D>>> (wavefunction_d, x_grid_d);

    genInfintesimalPropagator <<<grid_size, block_size>>> (infintesimal_propagator_d, x_grid_d);
    genAnalyticPropagator <<<grid_size, block_size>>> (analytic_propagator_d, x_grid_d, EPST);

    // Compute the finite step propagator that we will use.
    genPropagator (propagator_d, infintesimal_propagator_d, EPST_PER_STEP);

#if ENERGY
    unsigned int numSteps = 256*256;

    hipDoubleComplex *trace_d;
    hipMalloc( &trace_d, numSteps * sizeof(hipDoubleComplex) );

    hipDoubleComplex *trace_fft_d;
    hipMalloc( &trace_fft_d, numSteps * sizeof(hipDoubleComplex) );

    hipDoubleComplex *trace_fft_h;
    trace_fft_h = (hipDoubleComplex*) malloc( numSteps * sizeof(hipDoubleComplex) );

    //computeTrace( trace_d, analytic_propagator_d, numSteps);
    computeTrace( trace_d, infintesimal_propagator_d, numSteps);
    computeFFT( trace_fft_d, trace_d, numSteps, HIPFFT_BACKWARD );

    //hipMemcpy( x_grid_h, x_grid_d, NUM_D * sizeof(double), hipMemcpyDeviceToHost );
    //computeTraceFFTAnalytic( trace_fft_d, x_grid_h, numSteps);

    //fprintCudaVector( std::cout, trace_d, numSteps, "Trace" );
    //fprintCudaVector( std::cout, trace_fft_d, numSteps, "Trace FFT" );

    //computeStatistics(statistics, x_grid_d, wavefunction_d);


    hipMemcpy(trace_fft_h, trace_fft_d, numSteps*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    std::cout << std::setprecision(8);
    for(int i=0; i < numSteps; i++) {
        std::cout << std::setw(16) << ((double)i)*NUM_T/numSteps << std::setw(16) << hipCabs(trace_fft_h[i])/numSteps << std::endl;
    }
#elif OPENGL
    // Now set up the openGL and glut context and everything.
    initGL(argc, argv);
    // Set this option so that execution continues when the window is closed
    glutSetOption( GLUT_ACTION_ON_WINDOW_CLOSE, GLUT_ACTION_CONTINUE_EXECUTION);
    glutMainLoop();

#else
    std::ofstream outFile;
    outFile.open("statistics.out");
    // Or just run everything in our own loop to skip using openGL
    //printPropagator();

    outFile << std::setprecision(8);
    int i = 0;
    do {
        computeStatistics(statistics, x_grid_d, wavefunction_d);

        outFile << std::setw(16) << i*EPST*EPST_PER_STEP;
        for (int j=0; j < NUMSTATS; j++) {
            outFile << std::setw(16) << statistics[j];
        }
        outFile << std::endl;

        applyPropagator( wavefunction_d, propagator_d);
        i++;

    } while ( i*EPST*EPST_PER_STEP <= 8*2*PI);

    outFile << std::endl;

    outFile.close();

    /*
    std::ofstream analytic_file;
    analytic_file.open("analytic.csv");
    fprintCudaMatrix( analytic_file, analytic_propagator_d, NUM_D, NUM_D, "Analytic Propagator");
    analytic_file.close();

    std::ofstream propagator_file;
    propagator_file.open("propagator.csv");
    fprintCudaMatrix( propagator_file, infintesimal_propagator_d, NUM_D, NUM_D, "Infinitesimal Propagator");
    propagator_file.close();
    */
#endif
}

// vim: fdm=syntax : tags+=~/.vim/tags/cudacomplete,~/.vim/tags/glcomplete,~/.vim/tags/cula
